/*
* Copyright 1993-2017 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

#include <stdint.h>
#include <hip/hip_runtime.h>
#include <type_traits>

typedef enum ColorSpaceStandard {
    ColorSpaceStandard_BT709 = 1,
    ColorSpaceStandard_Unspecified = 2,
    ColorSpaceStandard_Reserved = 3,
    ColorSpaceStandard_FCC = 4,
    ColorSpaceStandard_BT470 = 5,
    ColorSpaceStandard_BT601 = 6,
    ColorSpaceStandard_SMPTE240M = 7,
    ColorSpaceStandard_YCgCo = 8,
    ColorSpaceStandard_BT2020 = 9,
    ColorSpaceStandard_BT2020C = 10
} ColorSpaceStandard;

__constant__ float matYuv2Rgb[3][3];
__constant__ float matRgb2Yuv[3][3];

void inline GetConstants(int iMatrix, float &wr, float &wb, int &black, int &white, int &max) {
    black = 16; white = 235;
    max = 255;

    switch (iMatrix)
    {
    case ColorSpaceStandard_BT709:
    default:
        wr = 0.2126f; wb = 0.0722f;
        break;

    case ColorSpaceStandard_FCC:
        wr = 0.30f; wb = 0.11f;
        break;

    case ColorSpaceStandard_BT470:
    case ColorSpaceStandard_BT601:
        wr = 0.2990f; wb = 0.1140f;
        break;

    case ColorSpaceStandard_SMPTE240M:
        wr = 0.212f; wb = 0.087f;
        break;

    case ColorSpaceStandard_BT2020:
    case ColorSpaceStandard_BT2020C:
        wr = 0.2627f; wb = 0.0593f;
        // 10-bit only
        black = 64 << 6; white = 940 << 6;
        max = (1 << 16) - 1;
        break;
    }
}

void SetMatYuv2Rgb(int iMatrix, hipStream_t stream) {
    float wr, wb;
    int black, white, max;
    GetConstants(iMatrix, wr, wb, black, white, max);
    float mat[3][3] = {
        1.0f, 0.0f, (1.0f - wr) / 0.5f,
        1.0f, -wb * (1.0f - wb) / 0.5f / (1 - wb - wr), -wr * (1 - wr) / 0.5f / (1 - wb - wr),
        1.0f, (1.0f - wb) / 0.5f, 0.0f,
    };
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            mat[i][j] = (float)(1.0 * max / (white - black) * mat[i][j]);
        }
    }
    hipMemcpyToSymbolAsync(HIP_SYMBOL(matYuv2Rgb), mat, sizeof(mat), 0, hipMemcpyHostToDevice, stream);
}

void SetMatRgb2Yuv(int iMatrix, hipStream_t stream) {
    float wr, wb;
    int black, white, max;
    GetConstants(iMatrix, wr, wb, black, white, max);
    float mat[3][3] = {
        wr, 1.0f - wb - wr, wb,
        -0.5f * wr / (1.0f - wb), -0.5f * (1 - wb - wr) / (1.0f - wb), 0.5f,
        0.5f, -0.5f * (1.0f - wb - wr) / (1.0f - wr), -0.5f * wb / (1.0f - wr),
    };
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            mat[i][j] = (float)(1.0 * (white - black) / max * mat[i][j]);
        }
    }
    hipMemcpyToSymbolAsync(HIP_SYMBOL(matRgb2Yuv), mat, sizeof(mat), 0, hipMemcpyHostToDevice, stream);
}

template<class T>
__device__ static T Clamp(T x, T lower, T upper) {
    return x < lower ? lower : (x > upper ? upper : x);
}

template<class Rgb, class YuvUnit>
__device__ inline Rgb YuvToRgbForPixel(YuvUnit y, YuvUnit u, YuvUnit v) {
    const int 
        low = 1 << (sizeof(YuvUnit) * 8 - 4),
        mid = 1 << (sizeof(YuvUnit) * 8 - 1);
    float fy = (int)y - low, fu = (int)u - mid, fv = (int)v - mid;
    const float maxf = (1 << sizeof(YuvUnit) * 8) - 1.0f;
    YuvUnit 
        r = (YuvUnit)Clamp(matYuv2Rgb[0][0] * fy + matYuv2Rgb[0][1] * fu + matYuv2Rgb[0][2] * fv, 0.0f, maxf),
        g = (YuvUnit)Clamp(matYuv2Rgb[1][0] * fy + matYuv2Rgb[1][1] * fu + matYuv2Rgb[1][2] * fv, 0.0f, maxf),
        b = (YuvUnit)Clamp(matYuv2Rgb[2][0] * fy + matYuv2Rgb[2][1] * fu + matYuv2Rgb[2][2] * fv, 0.0f, maxf);
    
    Rgb rgb{};
    const int nShift = abs((int)sizeof(YuvUnit) - (int)sizeof(rgb.c.r)) * 8;
    if (sizeof(YuvUnit) >= sizeof(rgb.c.r)) {
        rgb.c.r = r >> nShift;
        rgb.c.g = g >> nShift;
        rgb.c.b = b >> nShift;
    } else {
        rgb.c.r = r << nShift;
        rgb.c.g = g << nShift;
        rgb.c.b = b << nShift;
    }
    rgb.c.a = 255u;
    return rgb;
}

template<class YuvUnitx2, class Rgb, class RgbIntx2>
__global__ static void YuvToRgbKernel(uint8_t *pYuv, int nYuvPitch, uint8_t *pRgb, int nRgbPitch, int nWidth, int nHeight) {
    int x = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
    int y = (threadIdx.y + blockIdx.y * blockDim.y) * 2;
    if (x + 1 >= nWidth || y + 1 >= nHeight) {
        return;
    }

    uint8_t *pSrc = pYuv + x * sizeof(YuvUnitx2) / 2 + y * nYuvPitch;
    uint8_t *pDst = pRgb + x * sizeof(Rgb) + y * nRgbPitch;

    YuvUnitx2 l0 = *(YuvUnitx2 *)pSrc;
    YuvUnitx2 l1 = *(YuvUnitx2 *)(pSrc + nYuvPitch);
    YuvUnitx2 ch = *(YuvUnitx2 *)(pSrc + (nHeight - y / 2) * nYuvPitch);

    *(RgbIntx2 *)pDst = RgbIntx2 {
        YuvToRgbForPixel<Rgb>(l0.x, ch.x, ch.y).d, 
        YuvToRgbForPixel<Rgb>(l0.y, ch.x, ch.y).d,
    };
    *(RgbIntx2 *)(pDst + nRgbPitch) = RgbIntx2 {
        YuvToRgbForPixel<Rgb>(l1.x, ch.x, ch.y).d, 
        YuvToRgbForPixel<Rgb>(l1.y, ch.x, ch.y).d,
    };
}

template<class T0, class T1> 
__device__ T0 ToValue(T1 v) {
    if (std::is_same<T0, float>::value) {
        return v / (1.0f * ((1 << sizeof(v) * 8) - 1));
    }
    return v;
}

template<class YuvUnitx2, class Rgb, class RgbUnitx2>
__global__ static void YuvToRgbPlanarKernel(uint8_t *pYuv, int nYuvPitch, uint8_t *pRgbp, int nRgbpPitch, int nWidth, int nHeight) {
    int x = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
    int y = (threadIdx.y + blockIdx.y * blockDim.y) * 2;
    if (x + 1 >= nWidth || y + 1 >= nHeight) {
        return;
    }

    uint8_t *pSrc = pYuv + x * sizeof(YuvUnitx2) / 2 + y * nYuvPitch;

    YuvUnitx2 l0 = *(YuvUnitx2 *)pSrc;
    YuvUnitx2 l1 = *(YuvUnitx2 *)(pSrc + nYuvPitch);
    YuvUnitx2 ch = *(YuvUnitx2 *)(pSrc + (nHeight - y / 2) * nYuvPitch);

    Rgb rgb0 = YuvToRgbForPixel<Rgb>(l0.x, ch.x, ch.y),
        rgb1 = YuvToRgbForPixel<Rgb>(l0.y, ch.x, ch.y),
        rgb2 = YuvToRgbForPixel<Rgb>(l1.x, ch.x, ch.y),
        rgb3 = YuvToRgbForPixel<Rgb>(l1.y, ch.x, ch.y);

    uint8_t *pDst = pRgbp + x * sizeof(RgbUnitx2) / 2 + y * nRgbpPitch;
    *(RgbUnitx2 *)pDst                = RgbUnitx2 {ToValue<decltype(RgbUnitx2::x)>(rgb0.v.x), ToValue<decltype(RgbUnitx2::x)>(rgb1.v.x)};
    *(RgbUnitx2 *)(pDst + nRgbpPitch) = RgbUnitx2 {ToValue<decltype(RgbUnitx2::x)>(rgb2.v.x), ToValue<decltype(RgbUnitx2::x)>(rgb3.v.x)};
    pDst += nRgbpPitch * nHeight;
    *(RgbUnitx2 *)pDst                = RgbUnitx2 {ToValue<decltype(RgbUnitx2::x)>(rgb0.v.y), ToValue<decltype(RgbUnitx2::x)>(rgb1.v.y)};
    *(RgbUnitx2 *)(pDst + nRgbpPitch) = RgbUnitx2 {ToValue<decltype(RgbUnitx2::x)>(rgb2.v.y), ToValue<decltype(RgbUnitx2::x)>(rgb3.v.y)};
    pDst += nRgbpPitch * nHeight;
    *(RgbUnitx2 *)pDst                = RgbUnitx2 {ToValue<decltype(RgbUnitx2::x)>(rgb0.v.z), ToValue<decltype(RgbUnitx2::x)>(rgb1.v.z)};
    *(RgbUnitx2 *)(pDst + nRgbpPitch) = RgbUnitx2 {ToValue<decltype(RgbUnitx2::x)>(rgb2.v.z), ToValue<decltype(RgbUnitx2::x)>(rgb3.v.z)};
}

union BGRA32 {
    uint32_t d;
    uchar4 v;
    struct {
        uint8_t b, g, r, a;
    } c;
};

union RGBA32 {
    uint32_t d;
    uchar4 v;
    struct {
        uint8_t r, g, b, a;
    } c;
};

union BGRA64 {
    uint64_t d;
    ushort4 v;
    struct {
        uint16_t b, g, r, a;
    } c;
};

void Nv12ToBgra32(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix, hipStream_t stream) {
    SetMatYuv2Rgb(iMatrix, stream);
    YuvToRgbKernel<uchar2, BGRA32, uint2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpNv12, nNv12Pitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

void Nv12ToRgba32(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix, hipStream_t stream) {
    SetMatYuv2Rgb(iMatrix, stream);
    YuvToRgbKernel<uchar2, RGBA32, uint2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpNv12, nNv12Pitch, dpBgra, nBgraPitch, nWidth, nHeight);
}
void Nv12ToBgra64(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix, hipStream_t stream) {
    SetMatYuv2Rgb(iMatrix, stream);
    YuvToRgbKernel<uchar2, BGRA64, ulonglong2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpNv12, nNv12Pitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

void P016ToBgra32(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix, hipStream_t stream) {
    SetMatYuv2Rgb(iMatrix, stream);
    YuvToRgbKernel<ushort2, BGRA32, uint2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpP016, nP016Pitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

void P016ToBgra64(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix, hipStream_t stream) {
    SetMatYuv2Rgb(iMatrix, stream);
    YuvToRgbKernel<ushort2, BGRA64, ulonglong2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpP016, nP016Pitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

void Nv12ToBgrPlanar(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix, hipStream_t stream) {
    SetMatYuv2Rgb(iMatrix, stream);
    YuvToRgbPlanarKernel<uchar2, BGRA32, uchar2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpNv12, nNv12Pitch, dpBgrp, nBgrpPitch, nWidth, nHeight);
}

void Nv12ToRgbPlanar(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix, hipStream_t stream) {
    SetMatYuv2Rgb(iMatrix, stream);
    YuvToRgbPlanarKernel<uchar2, RGBA32, uchar2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpNv12, nNv12Pitch, dpBgrp, nBgrpPitch, nWidth, nHeight);
}
void P016ToBgrPlanar(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix, hipStream_t stream) {
    SetMatYuv2Rgb(iMatrix, stream);
    YuvToRgbPlanarKernel<ushort2, BGRA32, uchar2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpP016, nP016Pitch, dpBgrp, nBgrpPitch, nWidth, nHeight);
}

void Nv12ToBgrFloatPlanar(uint8_t *dpNv12, int nNv12Pitch, float *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix, hipStream_t stream) {
    SetMatYuv2Rgb(iMatrix, stream);
    YuvToRgbPlanarKernel<uchar2, BGRA32, float2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2), 0, stream>>>
        (dpNv12, nNv12Pitch, (uint8_t *)dpBgrp, nBgrpPitch, nWidth, nHeight);
}

void Nv12ToRgbFloatPlanar(uint8_t *dpNv12, int nNv12Pitch, float *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix, hipStream_t stream) {
    SetMatYuv2Rgb(iMatrix, stream);
    YuvToRgbPlanarKernel<uchar2, RGBA32, float2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2), 0, stream>>>
        (dpNv12, nNv12Pitch, (uint8_t *)dpBgrp, nBgrpPitch, nWidth, nHeight);
}

void P016ToBgrFloatPlanar(uint8_t *dpP016, int nP016Pitch, float *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix, hipStream_t stream) {
    SetMatYuv2Rgb(iMatrix, stream);
    YuvToRgbPlanarKernel<ushort2, BGRA32, float2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2), 0, stream>>>
        (dpP016, nP016Pitch, (uint8_t *)dpBgrp, nBgrpPitch, nWidth, nHeight);
}

template<class YuvUnit, class RgbUnit>
__device__ inline YuvUnit RgbToY(RgbUnit r, RgbUnit g, RgbUnit b) {
    const YuvUnit low = 1 << (sizeof(YuvUnit) * 8 - 4);
    return matRgb2Yuv[0][0] * r + matRgb2Yuv[0][1] * g + matRgb2Yuv[0][2] * b + low;
}

template<class YuvUnit, class RgbUnit>
__device__ inline YuvUnit RgbToU(RgbUnit r, RgbUnit g, RgbUnit b) {
    const YuvUnit mid = 1 << (sizeof(YuvUnit) * 8 - 1);
    return matRgb2Yuv[1][0] * r + matRgb2Yuv[1][1] * g + matRgb2Yuv[1][2] * b + mid;
}

template<class YuvUnit, class RgbUnit>
__device__ inline YuvUnit RgbToV(RgbUnit r, RgbUnit g, RgbUnit b) {
    const YuvUnit mid = 1 << (sizeof(YuvUnit) * 8 - 1);
    return matRgb2Yuv[2][0] * r + matRgb2Yuv[2][1] * g + matRgb2Yuv[2][2] * b + mid;
}

template<class YuvUnitx2, class Rgb, class RgbIntx2>
__global__ static void RgbToYuvKernel(uint8_t *pRgb, int nRgbPitch, uint8_t *pYuv, int nYuvPitch, int nWidth, int nHeight) {
    int x = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
    int y = (threadIdx.y + blockIdx.y * blockDim.y) * 2;
    if (x + 1 >= nWidth || y + 1 >= nHeight) {
        return;
    }

    uint8_t *pSrc = pRgb + x * sizeof(Rgb) + y * nRgbPitch;
    RgbIntx2 int2a = *(RgbIntx2 *)pSrc;
    RgbIntx2 int2b = *(RgbIntx2 *)(pSrc + nRgbPitch);

    Rgb rgb[4] = {int2a.x, int2a.y, int2b.x, int2b.y};
    decltype(Rgb::c.r)
        r = (rgb[0].c.r + rgb[1].c.r + rgb[2].c.r + rgb[3].c.r) / 4,
        g = (rgb[0].c.g + rgb[1].c.g + rgb[2].c.g + rgb[3].c.g) / 4,
        b = (rgb[0].c.b + rgb[1].c.b + rgb[2].c.b + rgb[3].c.b) / 4;

    uint8_t *pDst = pYuv + x * sizeof(YuvUnitx2) / 2 + y * nYuvPitch;
    *(YuvUnitx2 *)pDst = YuvUnitx2 {
        RgbToY<decltype(YuvUnitx2::x)>(rgb[0].c.r, rgb[0].c.g, rgb[0].c.b),
        RgbToY<decltype(YuvUnitx2::x)>(rgb[1].c.r, rgb[1].c.g, rgb[1].c.b),
    };
    *(YuvUnitx2 *)(pDst + nYuvPitch) = YuvUnitx2 {
        RgbToY<decltype(YuvUnitx2::x)>(rgb[2].c.r, rgb[2].c.g, rgb[2].c.b),
        RgbToY<decltype(YuvUnitx2::x)>(rgb[3].c.r, rgb[3].c.g, rgb[3].c.b),
    };
    *(YuvUnitx2 *)(pDst + (nHeight - y / 2) * nYuvPitch) = YuvUnitx2 {
        RgbToU<decltype(YuvUnitx2::x)>(r, g, b), 
        RgbToV<decltype(YuvUnitx2::x)>(r, g, b),
    };
}

void Bgra64ToP016(uint8_t *dpBgra, int nBgraPitch, uint8_t *dpP016, int nP016Pitch, int nWidth, int nHeight, int iMatrix, hipStream_t stream) {
    SetMatRgb2Yuv(iMatrix, stream);
    RgbToYuvKernel<ushort2, BGRA64, ulonglong2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpBgra, nBgraPitch, dpP016, nP016Pitch, nWidth, nHeight);
}
